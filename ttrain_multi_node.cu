#ifdef MULTI_GPU
#include <nccl.h>
#endif

typedef struct {
    int process_rank;
    int num_processes;
    int local_device_idx;

    int zero_stage;
    size_t shard_num_parameters;
#ifdef MULTI_GPU
    ncclComm_t nccl_comm;
    hipStream_t nccl_stream;
    hipEvent_t compute_nccl_sync;
    float* unified_buffer;
#endif
} MultiGpuConfig;

MultiGpuConfig multi_gpu_config_init() {
#ifdef MULTI_GPU
    MultiGpuConfig result;

}
int main(int argc, char* argv[]) {
    const char* train_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
    const char* val_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
    const char* load_filename = "gpt2_124M_bf16.bin";
    const char* lr_scheduler_type = "cosine";
    const char* output_log_dir = NULL;

    multi_gpu_config
}